// a cuda app.  we will convert this to opencl, and run it :-)

#include <iostream>
#include <memory>
#include <cassert>

using namespace std;

#include <hip/hip_runtime.h>

__global__ void setValue(float *data, int idx, float value) {
    if(threadIdx.x == 0) {
        data[idx] = value;
    }
}

int main(int argc, char *argv[]) {
    int N = 1024;

    float *gpuFloats;
    hipMalloc((void**)(&gpuFloats), N * sizeof(float));

    setValue<<<dim3(32, 1, 1), dim3(32, 1, 1)>>>(gpuFloats, 2, 123.0f);

    float hostFloats[4];
    hipMemcpy(hostFloats, gpuFloats, 4 * sizeof(float), hipMemcpyDeviceToHost);
    cout << "hostFloats[2] " << hostFloats[2] << endl;
    assert(hostFloats[2] == 123.0f);

    setValue<<<dim3(32, 1, 1), dim3(32, 1, 1)>>>(gpuFloats, 2, 222.0f);
    hipMemcpy(hostFloats, gpuFloats, 4 * sizeof(float), hipMemcpyDeviceToHost);
    cout << "hostFloats[2] " << hostFloats[2] << endl;
    assert(hostFloats[2] == 222.0f);

    hostFloats[2] = 444.0f;
    hipMemcpy(gpuFloats, hostFloats, 4 * sizeof(float), hipMemcpyHostToDevice);
    hostFloats[2] = 555.0f;
    hipMemcpy(hostFloats, gpuFloats, 4 * sizeof(float), hipMemcpyDeviceToHost);
    cout << "hostFloats[2] " << hostFloats[2] << endl;
    assert(hostFloats[2] == 444.0f);

    hipFree(gpuFloats);

    return 0;
}
