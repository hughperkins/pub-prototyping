// This is from Eigen unsupported/test/cxx11_tensor_cuda.cu

#define EIGEN_TEST_NO_LONGDOUBLE
#define EIGEN_TEST_NO_COMPLEX
#define EIGEN_TEST_FUNC cuda_elementwise
#define EIGEN_USE_GPU

// #if defined __CUDACC_VER__ && __CUDACC_VER__ >= 70500
// #include <hip/hip_fp16.h>
// #endif
#include <unsupported/Eigen/CXX11/Tensor>

#include "main.h"

#include <iostream>

using Eigen::Tensor;

void test_cuda_elementwise()
{
  Tensor<float, 3> in1(Eigen::array<Eigen::DenseIndex, 3>(72,53,97));
  Tensor<float, 3> in2(Eigen::array<Eigen::DenseIndex, 3>(72,53,97));
  // Tensor<float, 3> in3(Eigen::array<Eigen::DenseIndex, 3>(72,53,97));
  Tensor<bool, 3> out(Eigen::array<Eigen::DenseIndex, 3>(72,53,97));
  in1.setRandom();
  in2.setRandom();
  // in3.setRandom();

  std::size_t in1_bytes = in1.size() * sizeof(float);
  std::size_t in2_bytes = in2.size() * sizeof(float);
  // std::size_t in3_bytes = in3.size() * sizeof(float);
  std::size_t out_bytes = out.size() * sizeof(bool);

  float* d_in1;
  float* d_in2;
  // float* d_in3;
  bool* d_out;
  hipMalloc((void**)(&d_in1), in1_bytes);
  hipMalloc((void**)(&d_in2), in2_bytes);
  // hipMalloc((void**)(&d_in3), in3_bytes);
  hipMalloc((void**)(&d_out), out_bytes);

  hipMemcpy(d_in1, in1.data(), in1_bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_in2, in2.data(), in2_bytes, hipMemcpyHostToDevice);
  // hipMemcpy(d_in3, in3.data(), in3_bytes, hipMemcpyHostToDevice);

  Eigen::CudaStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<float, 3> > gpu_in1(d_in1, Eigen::array<Eigen::DenseIndex, 3>(72,53,97));
  Eigen::TensorMap<Eigen::Tensor<float, 3> > gpu_in2(d_in2, Eigen::array<Eigen::DenseIndex, 3>(72,53,97));
  // Eigen::TensorMap<Eigen::Tensor<float, 3> > gpu_in3(d_in3, Eigen::array<Eigen::DenseIndex, 3>(72,53,97));
  Eigen::TensorMap<Eigen::Tensor<bool, 3> > gpu_out(d_out, Eigen::array<Eigen::DenseIndex, 3>(72,53,97));

  gpu_out.device(gpu_device) = gpu_in1 == gpu_in2;

  assert(hipMemcpyAsync(out.data(), d_out, out_bytes, hipMemcpyDeviceToHost, gpu_device.stream()) == hipSuccess);
  assert(hipStreamSynchronize(gpu_device.stream()) == hipSuccess);

  for (int i = 0; i < 72; ++i) {
    for (int j = 0; j < 53; ++j) {
      for (int k = 0; k < 97; ++k) {
        assert(out(Eigen::array<Eigen::DenseIndex, 3>(i,j,k)) == (in1(Eigen::array<Eigen::DenseIndex, 3>(i,j,k)) == in2(Eigen::array<Eigen::DenseIndex, 3>(i,j,k))));
      }
    }
  }

  hipFree(d_in1);
  hipFree(d_in2);
  hipFree(d_out);
  std::cout << "finished ok" << std::endl;
}
