// This is from Eigen unsupported/test/cxx11_tensor_cuda.cu

#define EIGEN_TEST_NO_LONGDOUBLE
#define EIGEN_TEST_NO_COMPLEX
#define EIGEN_TEST_FUNC cuda_nullary
#define EIGEN_USE_GPU

// #if defined __CUDACC_VER__ && __CUDACC_VER__ >= 70500
// #include <hip/hip_fp16.h>
// #endif
#include <unsupported/Eigen/CXX11/Tensor>

#include "main.h"

#include <iostream>

using Eigen::Tensor;

void test_cuda_nullary() {
  Tensor<float, 1, 0, int> in1(2);
  Tensor<float, 1, 0, int> in2(2);
  in1.setRandom();
  in2.setRandom();

  std::size_t tensor_bytes = in1.size() * sizeof(float);

  float* d_in1;
  float* d_in2;
  hipMalloc((void**)(&d_in1), tensor_bytes);
  hipMalloc((void**)(&d_in2), tensor_bytes);
  hipMemcpy(d_in1, in1.data(), tensor_bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_in2, in2.data(), tensor_bytes, hipMemcpyHostToDevice);

  Eigen::CudaStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<float, 1, 0, int>, Eigen::Aligned> gpu_in1(
      d_in1, 2);
  Eigen::TensorMap<Eigen::Tensor<float, 1, 0, int>, Eigen::Aligned> gpu_in2(
      d_in2, 2);

  gpu_in1.device(gpu_device) = gpu_in1.constant(3.14f);
  gpu_in2.device(gpu_device) = gpu_in2.random();

  Tensor<float, 1, 0, int> new1(2);
  Tensor<float, 1, 0, int> new2(2);

  assert(hipMemcpyAsync(new1.data(), d_in1, tensor_bytes, hipMemcpyDeviceToHost,
                         gpu_device.stream()) == hipSuccess);
  assert(hipMemcpyAsync(new2.data(), d_in2, tensor_bytes, hipMemcpyDeviceToHost,
                         gpu_device.stream()) == hipSuccess);

  std::cout << "1" << std::endl;
  assert(hipStreamSynchronize(gpu_device.stream()) == hipSuccess);
  std::cout << "2" << std::endl;

  for (int i = 0; i < 2; ++i) {
    VERIFY_IS_APPROX(new1(i), 3.14f);
    VERIFY_IS_NOT_EQUAL(new2(i), in2(i));
  }

  hipFree(d_in1);
  hipFree(d_in2);
}
