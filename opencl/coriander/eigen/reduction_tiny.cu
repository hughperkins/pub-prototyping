// This is from Eigen unsupported/test/cxx11_tensor_cuda.cu

#define EIGEN_TEST_NO_LONGDOUBLE
#define EIGEN_TEST_NO_COMPLEX
#define EIGEN_TEST_FUNC cuda_reduction
#define EIGEN_USE_GPU

// #if defined __CUDACC_VER__ && __CUDACC_VER__ >= 70500
// #include <hip/hip_fp16.h>
// #endif
#include <unsupported/Eigen/CXX11/Tensor>

#include "main.h"

#include <iostream>

using Eigen::Tensor;

void test_cuda_reduction()
{
  const int N = 3;

  Tensor<float, 1> in1(N);
  Tensor<float, 1> out(1);
  in1.setRandom();

  std::size_t in1_bytes = in1.size() * sizeof(float);
  std::size_t out_bytes = out.size() * sizeof(float);

  std::cout << "in1.data(): ";
  for(int i = 0; i < N; i++) {
    std::cout << " " << in1.data()[i];
  }
  std::cout << std::endl;

  float* d_in1;
  float* d_out;
  hipMalloc((void**)(&d_in1), in1_bytes);
  hipMalloc((void**)(&d_out), out_bytes);

  hipMemcpy(d_in1, in1.data(), in1_bytes, hipMemcpyHostToDevice);

  Eigen::CudaStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<float, 1> > gpu_in1(d_in1, N);
  Eigen::TensorMap<Eigen::Tensor<float, 1> > gpu_out(d_out, 1);

  array<Eigen::DenseIndex, 1> reduction_axis;
  reduction_axis[0] = 0;

  gpu_out.device(gpu_device) = gpu_in1.sum(reduction_axis);

  assert(hipMemcpyAsync(out.data(), d_out, out_bytes, hipMemcpyDeviceToHost, gpu_device.stream()) == hipSuccess);
  assert(hipStreamSynchronize(gpu_device.stream()) == hipSuccess);

  std::cout << "out.data()[0] " << out.data()[0] << std::endl;

  assert(hipMemcpyAsync(in1.data(), d_in1, in1_bytes, hipMemcpyDeviceToHost, gpu_device.stream()) == hipSuccess);
  assert(hipStreamSynchronize(gpu_device.stream()) == hipSuccess);
  std::cout << "in1.data()[0] " << in1.data()[0] << std::endl;

  float sum = 0;
  for (int i = 0; i < N; ++i) {
    sum += in1(i);
  }
  std::cout << "expected sum: " << sum << std:: endl;
  std::cout << "kernel sum: " << out(0) << std::endl;
  VERIFY_IS_APPROX(out(0), sum);

  hipFree(d_in1);
  hipFree(d_out);
}
