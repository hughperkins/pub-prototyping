#include "hip/hip_runtime.h"
// This is from Eigen unsupported/test/cxx11_tensor_cuda.cu

#define EIGEN_TEST_NO_LONGDOUBLE
#define EIGEN_TEST_NO_COMPLEX
#define EIGEN_TEST_FUNC cuda_elementwise
#define EIGEN_USE_GPU

// #if defined __CUDACC_VER__ && __CUDACC_VER__ >= 70500
// #include <hip/hip_fp16.h>
// #endif
#include <unsupported/Eigen/CXX11/Tensor>

#include "main.h"

#include <iostream>

using Eigen::Tensor;

void test_cuda_elementwise()
{
  Tensor<float, 3> in1(Eigen::array<Eigen::DenseIndex, 3>(72,53,97));
  Tensor<float, 3> out(Eigen::array<Eigen::DenseIndex, 3>(72,53,97));
  in1.setRandom();

  std::size_t in1_bytes = in1.size() * sizeof(float);
  std::size_t out_bytes = out.size() * sizeof(float);

  float* d_in1;
  float* d_out;
  hipMalloc((void**)(&d_in1), in1_bytes);
  hipMalloc((void**)(&d_out), out_bytes);

  hipMemcpy(d_in1, in1.data(), in1_bytes, hipMemcpyHostToDevice);

  Eigen::CudaStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<float, 3> > gpu_in1(d_in1, Eigen::array<Eigen::DenseIndex, 3>(72,53,97));
  Eigen::TensorMap<Eigen::Tensor<float, 3> > gpu_out(d_out, Eigen::array<Eigen::DenseIndex, 3>(72,53,97));

  gpu_out.device(gpu_device) = (gpu_in1 - 0.5f).abs();

  assert(hipMemcpyAsync(out.data(), d_out, out_bytes, hipMemcpyDeviceToHost, gpu_device.stream()) == hipSuccess);
  assert(hipStreamSynchronize(gpu_device.stream()) == hipSuccess);

  for (int i = 0; i < 72; ++i) {
    for (int j = 0; j < 53; ++j) {
      for (int k = 0; k < 97; ++k) {
        float actual = out(Eigen::array<Eigen::DenseIndex, 3>(i,j,k));
        float expected = std::abs(in1(Eigen::array<Eigen::DenseIndex, 3>(i,j,k))- 0.5f);
        float diff = actual - expected;
        if(std::abs(diff) >= 0.00001f) {
          std::cout << "i=" << i << " j=" << j << " k=" << k << " diff=" << diff << std::endl;
        }
        VERIFY_IS_APPROX(actual, expected);
      }
    }
  }

  hipFree(d_in1);
  hipFree(d_out);
  std::cout << "finished ok" << std::endl;
}
