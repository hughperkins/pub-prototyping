// This is from Eigen unsupported/test/cxx11_tensor_cuda.cu

#define EIGEN_TEST_NO_LONGDOUBLE
#define EIGEN_TEST_NO_COMPLEX
#define EIGEN_TEST_FUNC cuda_elementwise_small
#define EIGEN_USE_GPU

// #if defined __CUDACC_VER__ && __CUDACC_VER__ >= 70500
// #include <hip/hip_fp16.h>
// #endif
#include <unsupported/Eigen/CXX11/Tensor>

#include "main.h"

using Eigen::Tensor;

// #include "main.h"

void test_cuda_elementwise_small() {
  Tensor<float, 1> in1(Eigen::array<Eigen::DenseIndex, 1>(2));
  Tensor<float, 1> in2(Eigen::array<Eigen::DenseIndex, 1>(2));
  Tensor<float, 1> out(Eigen::array<Eigen::DenseIndex, 1>(2));
  in1.setRandom();
  in2.setRandom();

  std::cout << "in1 " << in1 << std::endl;
  std::cout << "in2 " << in2 << std::endl;

  std::size_t in1_bytes = in1.size() * sizeof(float);
  std::size_t in2_bytes = in2.size() * sizeof(float);
  std::size_t out_bytes = out.size() * sizeof(float);

  float* d_in1;
  float* d_in2;
  float* d_out;
  hipMalloc((void**)(&d_in1), in1_bytes);
  hipMalloc((void**)(&d_in2), in2_bytes);
  hipMalloc((void**)(&d_out), out_bytes);

  hipMemcpy(d_in1, in1.data(), in1_bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_in2, in2.data(), in2_bytes, hipMemcpyHostToDevice);

  Eigen::CudaStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<float, 1>, Eigen::Aligned> gpu_in1(
       d_in1, Eigen::array<Eigen::DenseIndex, 1>(2));
  Eigen::TensorMap<Eigen::Tensor<float, 1>, Eigen::Aligned> gpu_in2(
       d_in2, Eigen::array<Eigen::DenseIndex, 1>(2));
  Eigen::TensorMap<Eigen::Tensor<float, 1>, Eigen::Aligned> gpu_out(
       d_out, Eigen::array<Eigen::DenseIndex, 1>(2));

  gpu_out.device(gpu_device) = gpu_in1 + gpu_in2;

  assert(hipMemcpyAsync(out.data(), d_out, out_bytes, hipMemcpyDeviceToHost,
                         gpu_device.stream()) == hipSuccess);
  assert(hipStreamSynchronize(gpu_device.stream()) == hipSuccess);

  for (int i = 0; i < 2; ++i) {
    VERIFY_IS_APPROX(
        out(Eigen::array<Eigen::DenseIndex, 1>(i)),
        in1(Eigen::array<Eigen::DenseIndex, 1>(i)) + in2(Eigen::array<Eigen::DenseIndex, 1>(i)));
  }

  hipFree(d_in1);
  hipFree(d_in2);
  hipFree(d_out);
}
