#define EIGEN_TEST_NO_LONGDOUBLE
#define EIGEN_TEST_NO_COMPLEX
#define EIGEN_TEST_FUNC cuda_reduction
#define EIGEN_USE_GPU

#include <unsupported/Eigen/CXX11/Tensor>

#include "main.h"

#include <iostream>

using Eigen::Tensor;

void test_cuda_reduction()
{
  Tensor<float, 4> in1(72,53,97,113);
  Tensor<float, 2> out(72,97);
  in1.setRandom();

  std::size_t in1_bytes = in1.size() * sizeof(float);
  std::size_t out_bytes = out.size() * sizeof(float);

  float* d_in1;
  float* d_out;
  hipMalloc((void**)(&d_in1), in1_bytes);
  hipMalloc((void**)(&d_out), out_bytes);

  hipMemcpy(d_in1, in1.data(), in1_bytes, hipMemcpyHostToDevice);

  Eigen::CudaStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<float, 4> > gpu_in1(d_in1, 72,53,97,113);
  Eigen::TensorMap<Eigen::Tensor<float, 2> > gpu_out(d_out, 72,97);

  array<Eigen::DenseIndex, 2> reduction_axis;
  reduction_axis[0] = 1;
  reduction_axis[1] = 3;

  gpu_out.device(gpu_device) = gpu_in1.mean(reduction_axis);

  assert(hipMemcpyAsync(out.data(), d_out, out_bytes, hipMemcpyDeviceToHost, gpu_device.stream()) == hipSuccess);
  assert(hipStreamSynchronize(gpu_device.stream()) == hipSuccess);

  for (int i = 0; i < 72; ++i) {
    for (int j = 0; j < 97; ++j) {
      float sum = 0;
      int count = 0;
      for (int k = 0; k < 53; ++k) {
        for (int l = 0; l < 113; ++l) {
          sum += in1(i, k, j, l);
          count++;
        }
      }
      float mean = sum / count;
      VERIFY_IS_APPROX(out(i,j), mean);
    }
  }

  hipFree(d_in1);
  hipFree(d_out);
}
