#include "hip/hip_runtime.h"
// Basically lifted straight from test/cxx11_tensor_argmax_cuda.cu

#define EIGEN_TEST_NO_LONGDOUBLE
#define EIGEN_TEST_NO_COMPLEX
#define EIGEN_TEST_FUNC argmax_all
#define EIGEN_USE_GPU

// this is needed on Beignet 1.2.1, Intel HD5500 (as far as I can tell?)
// #define EIGEN_DEFAULT_DENSE_INDEX_TYPE int32_t

// #define CALL_SUBTEST_1(expr) expr
// #define CALL_SUBTEST_2(expr) expr
// #define CALL_SUBTEST_3(expr) expr

#include <unsupported/Eigen/CXX11/Tensor>

#include "main.h"

#include <iostream>

using Eigen::Tensor;

void test_cuda_simple_argmax_tiny_rowmajor()
{
  std::cout << "test" << std::endl;
  #define N 5
  Tensor<float, 1, RowMajor> in(Eigen::array<DenseIndex, 1>(N));
  Tensor<DenseIndex, 1, RowMajor> out_max(Eigen::array<DenseIndex, 1>(1));

  in(0) = 3.0f;
  in(1) = 4.0f;
  in(2) = 7.0f;
  in(3) = 5.0f;
  in(4) = 1.0f;

  std::size_t in_bytes = in.size() * sizeof(float);
  std::size_t out_bytes = out_max.size() * sizeof(DenseIndex);

  float* d_in;
  DenseIndex* d_out_max;

  hipMalloc((void**)(&d_in), in_bytes);
  hipMalloc((void**)(&d_out_max), out_bytes);


  hipMemcpy(d_in, in.data(), in_bytes, hipMemcpyHostToDevice);

  Eigen::CudaStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<float, 1, RowMajor>, Aligned > gpu_in(d_in, Eigen::array<DenseIndex, 1>(N));
  Eigen::TensorMap<Eigen::Tensor<DenseIndex, 1, RowMajor>, Aligned > gpu_out_max(d_out_max, Eigen::array<DenseIndex, 1>(1));

  gpu_out_max.device(gpu_device) = gpu_in.argmax();

  assert(hipMemcpyAsync(out_max.data(), d_out_max, out_bytes, hipMemcpyDeviceToHost, gpu_device.stream()) == hipSuccess);
  assert(hipStreamSynchronize(gpu_device.stream()) == hipSuccess);

  VERIFY_IS_EQUAL(out_max(Eigen::array<DenseIndex, 1>(0)), 2);

  hipFree(d_in);
  hipFree(d_out_max);
}

void test_cuda_simple_argmax_rowmajor()
{
  std::cout << "test" << std::endl;
  Tensor<float, 3, RowMajor> in(Eigen::array<DenseIndex, 3>(72,53,97));
  Tensor<DenseIndex, 1, RowMajor> out_max(Eigen::array<DenseIndex, 1>(1));
  in.setRandom();
  in *= in.constant(100.0);
  in(21, 29, 76) = -1000.0;
  in(37, 43, 88) = 1000.0;

  std::size_t in_bytes = in.size() * sizeof(float);
  std::size_t out_bytes = out_max.size() * sizeof(DenseIndex);

  float* d_in;
  DenseIndex* d_out_max;
  hipMalloc((void**)(&d_in), in_bytes);
  hipMalloc((void**)(&d_out_max), out_bytes);

  hipMemcpy(d_in, in.data(), in_bytes, hipMemcpyHostToDevice);

  Eigen::CudaStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<float, 3, RowMajor>, Aligned > gpu_in(d_in, Eigen::array<DenseIndex, 3>(72,53,97));
  Eigen::TensorMap<Eigen::Tensor<DenseIndex, 1, RowMajor>, Aligned > gpu_out_max(d_out_max, Eigen::array<DenseIndex, 1>(1));

  gpu_out_max.device(gpu_device) = gpu_in.argmax();

  assert(hipMemcpyAsync(out_max.data(), d_out_max, out_bytes, hipMemcpyDeviceToHost, gpu_device.stream()) == hipSuccess);
  assert(hipStreamSynchronize(gpu_device.stream()) == hipSuccess);

  VERIFY_IS_EQUAL(out_max(Eigen::array<DenseIndex, 1>(0)), 37*53*97 + 43*97 + 88);

  hipFree(d_in);
  hipFree(d_out_max);
}

void test_cuda_simple_argmin_rowmajor()
{
  std::cout << "test" << std::endl;
  Tensor<float, 3, RowMajor> in(Eigen::array<DenseIndex, 3>(72,53,97));
  Tensor<DenseIndex, 1, RowMajor> out_min(Eigen::array<DenseIndex, 1>(1));
  in.setRandom();
  in *= in.constant(100.0);
  in(21, 29, 76) = -1000.0;
  in(37, 43, 88) = 1000.0;

  std::size_t in_bytes = in.size() * sizeof(float);
  std::size_t out_bytes = out_min.size() * sizeof(DenseIndex);

  float* d_in;
  DenseIndex* d_out_min;
  hipMalloc((void**)(&d_in), in_bytes);
  hipMalloc((void**)(&d_out_min), out_bytes);

  hipMemcpy(d_in, in.data(), in_bytes, hipMemcpyHostToDevice);

  Eigen::CudaStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<float, 3, RowMajor>, Aligned > gpu_in(d_in, Eigen::array<DenseIndex, 3>(72,53,97));
  Eigen::TensorMap<Eigen::Tensor<DenseIndex, 1, RowMajor>, Aligned > gpu_out_min(d_out_min, Eigen::array<DenseIndex, 1>(1));

  gpu_out_min.device(gpu_device) = gpu_in.argmin();

  assert(hipMemcpyAsync(out_min.data(), d_out_min, out_bytes, hipMemcpyDeviceToHost, gpu_device.stream()) == hipSuccess);
  assert(hipStreamSynchronize(gpu_device.stream()) == hipSuccess);

  VERIFY_IS_EQUAL(out_min(Eigen::array<DenseIndex, 1>(0)), 21*53*97 + 29*97 + 76);

  hipFree(d_in);
  hipFree(d_out_min);
}

void test_cuda_simple_argmax_colmajor()
{
  std::cout << "test" << std::endl;
  Tensor<float, 3, ColMajor> in(Eigen::array<DenseIndex, 3>(72,53,97));
  Tensor<DenseIndex, 1, ColMajor> out_max(Eigen::array<DenseIndex, 1>(1));
  in.setRandom();
  in *= in.constant(100.0);
  in(21, 29, 76) = -1000.0;
  in(37, 43, 88) = 1000.0;

  std::size_t in_bytes = in.size() * sizeof(float);
  std::size_t out_bytes = out_max.size() * sizeof(DenseIndex);

  float* d_in;
  DenseIndex* d_out_max;
  hipMalloc((void**)(&d_in), in_bytes);
  hipMalloc((void**)(&d_out_max), out_bytes);

  hipMemcpy(d_in, in.data(), in_bytes, hipMemcpyHostToDevice);

  Eigen::CudaStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<float, 3, ColMajor>, Aligned > gpu_in(d_in, Eigen::array<DenseIndex, 3>(72,53,97));
  Eigen::TensorMap<Eigen::Tensor<DenseIndex, 1, ColMajor>, Aligned > gpu_out_max(d_out_max, Eigen::array<DenseIndex, 1>(1));

  gpu_out_max.device(gpu_device) = gpu_in.argmax();

  assert(hipMemcpyAsync(out_max.data(), d_out_max, out_bytes, hipMemcpyDeviceToHost, gpu_device.stream()) == hipSuccess);
  assert(hipStreamSynchronize(gpu_device.stream()) == hipSuccess);

  VERIFY_IS_EQUAL(out_max(Eigen::array<DenseIndex, 1>(0)), 88*53*72 + 43*72 + 37);

  hipFree(d_in);
  hipFree(d_out_max);
}

void test_cuda_simple_argmin_colmajor()
{
  std::cout << "test" << std::endl;
  Tensor<float, 3, ColMajor> in(Eigen::array<DenseIndex, 3>(72,53,97));
  Tensor<DenseIndex, 1, ColMajor> out_min(Eigen::array<DenseIndex, 1>(1));
  in.setRandom();
  in *= in.constant(100.0);
  in(21, 29, 76) = -1000.0;
  in(37, 43, 88) = 1000.0;

  std::size_t in_bytes = in.size() * sizeof(float);
  std::size_t out_bytes = out_min.size() * sizeof(DenseIndex);

  float* d_in;
  DenseIndex* d_out_min;
  hipMalloc((void**)(&d_in), in_bytes);
  hipMalloc((void**)(&d_out_min), out_bytes);

  hipMemcpy(d_in, in.data(), in_bytes, hipMemcpyHostToDevice);

  Eigen::CudaStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<float, 3, ColMajor>, Aligned > gpu_in(d_in, Eigen::array<DenseIndex, 3>(72,53,97));
  Eigen::TensorMap<Eigen::Tensor<DenseIndex, 1, ColMajor>, Aligned > gpu_out_min(d_out_min, Eigen::array<DenseIndex, 1>(1));

  gpu_out_min.device(gpu_device) = gpu_in.argmin();

  assert(hipMemcpyAsync(out_min.data(), d_out_min, out_bytes, hipMemcpyDeviceToHost, gpu_device.stream()) == hipSuccess);
  assert(hipStreamSynchronize(gpu_device.stream()) == hipSuccess);

  VERIFY_IS_EQUAL(out_min(Eigen::array<DenseIndex, 1>(0)), 76*53*72 + 29*72 + 21);

  hipFree(d_in);
  hipFree(d_out_min);
}

template <int DataLayout>
void test_cuda_argmax_dim()
{
  Tensor<float, 4, DataLayout> tensor(2,3,5,7);
  std::vector<int> dims;
  dims.push_back(2); dims.push_back(3); dims.push_back(5); dims.push_back(7);

  for (int dim = 0; dim < 4; ++dim) {
    tensor.setRandom();
    tensor = (tensor + tensor.constant(0.5)).log();

    array<DenseIndex, 3> out_shape;
    for (int d = 0; d < 3; ++d) out_shape[d] = (d < dim) ? dims[d] : dims[d+1];

    Tensor<DenseIndex, 3, DataLayout> tensor_arg(out_shape);

    array<DenseIndex, 4> ix;
    for (int i = 0; i < 2; ++i) {
      for (int j = 0; j < 3; ++j) {
        for (int k = 0; k < 5; ++k) {
          for (int l = 0; l < 7; ++l) {
            ix[0] = i; ix[1] = j; ix[2] = k; ix[3] = l;
            if (ix[dim] != 0) continue;
            // suppose dim == 1, then for all i, k, l, set tensor(i, 0, k, l) = 10.0
            tensor(ix) = 10.0;
          }
        }
      }
    }

    std::size_t in_bytes = tensor.size() * sizeof(float);
    std::size_t out_bytes = tensor_arg.size() * sizeof(DenseIndex);

    float* d_in;
    DenseIndex* d_out;
    hipMalloc((void**)(&d_in), in_bytes);
    hipMalloc((void**)(&d_out), out_bytes);

    hipMemcpy(d_in, tensor.data(), in_bytes, hipMemcpyHostToDevice);

    Eigen::CudaStreamDevice stream;
    Eigen::GpuDevice gpu_device(&stream);

    Eigen::TensorMap<Eigen::Tensor<float, 4, DataLayout>, Aligned > gpu_in(d_in, Eigen::array<DenseIndex, 4>(2, 3, 5, 7));
    Eigen::TensorMap<Eigen::Tensor<DenseIndex, 3, DataLayout>, Aligned > gpu_out(d_out, out_shape);

    gpu_out.device(gpu_device) = gpu_in.argmax(dim);

    assert(hipMemcpyAsync(tensor_arg.data(), d_out, out_bytes, hipMemcpyDeviceToHost, gpu_device.stream()) == hipSuccess);
    assert(hipStreamSynchronize(gpu_device.stream()) == hipSuccess);

    VERIFY_IS_EQUAL(tensor_arg.size(),
                    size_t(2*3*5*7 / tensor.dimension(dim)));

    for (DenseIndex n = 0; n < tensor_arg.size(); ++n) {
      // Expect max to be in the first index of the reduced dimension
      VERIFY_IS_EQUAL(tensor_arg.data()[n], 0);
    }

    for (int i = 0; i < 2; ++i) {
      for (int j = 0; j < 3; ++j) {
        for (int k = 0; k < 5; ++k) {
          for (int l = 0; l < 7; ++l) {
            ix[0] = i; ix[1] = j; ix[2] = k; ix[3] = l;
            if (ix[dim] != tensor.dimension(dim) - 1) continue;
            // suppose dim == 1, then for all i, k, l, set tensor(i, 2, k, l) = 20.0
            tensor(ix) = 20.0;
          }
        }
      }
    }

    hipMemcpy(d_in, tensor.data(), in_bytes, hipMemcpyHostToDevice);

    gpu_out.device(gpu_device) = gpu_in.argmax(dim);

    assert(hipMemcpyAsync(tensor_arg.data(), d_out, out_bytes, hipMemcpyDeviceToHost, gpu_device.stream()) == hipSuccess);
    assert(hipStreamSynchronize(gpu_device.stream()) == hipSuccess);

    for (DenseIndex n = 0; n < tensor_arg.size(); ++n) {
      // Expect max to be in the last index of the reduced dimension
      VERIFY_IS_EQUAL(tensor_arg.data()[n], tensor.dimension(dim) - 1);
    }

    hipFree(d_in);
    hipFree(d_out);
  }
}

template <int DataLayout>
void test_cuda_argmin_dim()
{
  Tensor<float, 4, DataLayout> tensor(2,3,5,7);
  std::vector<int> dims;
  dims.push_back(2); dims.push_back(3); dims.push_back(5); dims.push_back(7);

  for (int dim = 0; dim < 4; ++dim) {
    tensor.setRandom();
    tensor = (tensor + tensor.constant(0.5)).log();

    array<DenseIndex, 3> out_shape;
    for (int d = 0; d < 3; ++d) out_shape[d] = (d < dim) ? dims[d] : dims[d+1];

    Tensor<DenseIndex, 3, DataLayout> tensor_arg(out_shape);

    array<DenseIndex, 4> ix;
    for (int i = 0; i < 2; ++i) {
      for (int j = 0; j < 3; ++j) {
        for (int k = 0; k < 5; ++k) {
          for (int l = 0; l < 7; ++l) {
            ix[0] = i; ix[1] = j; ix[2] = k; ix[3] = l;
            if (ix[dim] != 0) continue;
            // suppose dim == 1, then for all i, k, l, set tensor(i, 0, k, l) = 10.0
            tensor(ix) = -10.0;
          }
        }
      }
    }

    std::size_t in_bytes = tensor.size() * sizeof(float);
    std::size_t out_bytes = tensor_arg.size() * sizeof(DenseIndex);

    float* d_in;
    DenseIndex* d_out;
    hipMalloc((void**)(&d_in), in_bytes);
    hipMalloc((void**)(&d_out), out_bytes);

    hipMemcpy(d_in, tensor.data(), in_bytes, hipMemcpyHostToDevice);

    Eigen::CudaStreamDevice stream;
    Eigen::GpuDevice gpu_device(&stream);

    Eigen::TensorMap<Eigen::Tensor<float, 4, DataLayout>, Aligned > gpu_in(d_in, Eigen::array<DenseIndex, 4>(2, 3, 5, 7));
    Eigen::TensorMap<Eigen::Tensor<DenseIndex, 3, DataLayout>, Aligned > gpu_out(d_out, out_shape);

    gpu_out.device(gpu_device) = gpu_in.argmin(dim);

    assert(hipMemcpyAsync(tensor_arg.data(), d_out, out_bytes, hipMemcpyDeviceToHost, gpu_device.stream()) == hipSuccess);
    assert(hipStreamSynchronize(gpu_device.stream()) == hipSuccess);

    VERIFY_IS_EQUAL(tensor_arg.size(),
                    2*3*5*7 / tensor.dimension(dim));

    for (DenseIndex n = 0; n < tensor_arg.size(); ++n) {
      // Expect min to be in the first index of the reduced dimension
      VERIFY_IS_EQUAL(tensor_arg.data()[n], 0);
    }

    for (int i = 0; i < 2; ++i) {
      for (int j = 0; j < 3; ++j) {
        for (int k = 0; k < 5; ++k) {
          for (int l = 0; l < 7; ++l) {
            ix[0] = i; ix[1] = j; ix[2] = k; ix[3] = l;
            if (ix[dim] != tensor.dimension(dim) - 1) continue;
            // suppose dim == 1, then for all i, k, l, set tensor(i, 2, k, l) = 20.0
            tensor(ix) = -20.0;
          }
        }
      }
    }

    hipMemcpy(d_in, tensor.data(), in_bytes, hipMemcpyHostToDevice);

    gpu_out.device(gpu_device) = gpu_in.argmin(dim);

    assert(hipMemcpyAsync(tensor_arg.data(), d_out, out_bytes, hipMemcpyDeviceToHost, gpu_device.stream()) == hipSuccess);
    assert(hipStreamSynchronize(gpu_device.stream()) == hipSuccess);

    for (DenseIndex n = 0; n < tensor_arg.size(); ++n) {
      // Expect max to be in the last index of the reduced dimension
      VERIFY_IS_EQUAL(tensor_arg.data()[n], tensor.dimension(dim) - 1);
    }

    hipFree(d_in);
    hipFree(d_out);
  }
}

void test_argmax_all()
{
  std::cout << "test_argmax_all()" << std::endl;
  test_cuda_simple_argmax_tiny_rowmajor();
  test_cuda_simple_argmax_rowmajor();
  test_cuda_simple_argmin_rowmajor();
  test_cuda_simple_argmax_colmajor();
  test_cuda_simple_argmin_colmajor();

  test_cuda_argmax_dim<RowMajor>();
  test_cuda_argmax_dim<ColMajor>();
  test_cuda_argmin_dim<RowMajor>();
  test_cuda_argmin_dim<ColMajor>();
}
